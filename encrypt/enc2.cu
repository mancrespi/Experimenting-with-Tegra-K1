
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define THREADS_PER_BLOCK 1024
#define THRESHOLD 67108864

__global__ void encrypt(int n, char *m, char *k, char *c){
	int j, i =  blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n){
		for(j = 1; j <= 100; j++){
			 c[i] = m[i] ^ k[i];
		}
	}
}

int main(){
	/* Serial on Host */
	FILE *fp_m, *fp_k;
	char *m, *k, *c, ch; /* Host copies */
	char *d_m, *d_k, *d_c; /* Device copies */
	int s_m, s_k, j,i;
	float msecPerencrypt, msecTotal = 0.0f;
	//printf("setup ...\n");
	//get size of files to malloc data
	if (!(fp_m = fopen("../../file.txt", "r")))
		perror("failed to read message file\n");
	
	while( fscanf(fp_m,"%c",&ch) != EOF ){
		s_m++;
	}

	if (!(fp_k = fopen("../../key.txt", "r")))
		perror("failed to read key\n");
	
	while( fscanf(fp_k,"%c",&ch) != EOF ){
		s_k++;
	}
	
	//printf("mallocs cpu...\n");

	//malloc space for m, k, c
	if ( !(m = (char *)malloc(sizeof(char)*s_m)) ){ 
		printf("Failed on malloc for m\n");
		exit(EXIT_FAILURE);
	}

	if ( !(k = (char *)malloc(sizeof(char)*s_k)) ){ 
		printf("Failed on malloc for k\n");
		exit(EXIT_FAILURE);
	}

	if ( !(c = (char *)malloc(sizeof(char)*s_m)) ){ 
		printf("Failed on malloc for c\n");
		exit(EXIT_FAILURE);
	}

	/* Alloc space for device copies of m, k, c */
	hipError_t error;
	printf("mallocs gpu...\n");
	error = hipMalloc((void **)&d_m, s_m);
	error = hipMalloc((void **)&d_k, s_k);
	error = hipMalloc((void **)&d_c, s_m);

	fseek(fp_m, 0, 0);
	fseek(fp_k, 0, 0);

	//read into buffers
	printf("read data...\n");
	for( j = 0; fscanf(fp_m,"%c",&ch) != EOF; j++ ){
		m[j] = ch;
	}
	for( j = 0; fscanf(fp_k,"%c",&ch) != EOF; j++ ){
		k[j] = ch;
	}

	/* Copy inputs to device */
	printf("Copy to device...\n");
 	hipMemcpy(d_m, m, s_m, hipMemcpyHostToDevice);
 	hipMemcpy(d_k, k, s_k, hipMemcpyHostToDevice);
	
	printf("Setting up streams...\n");
	int sections = s_m/THRESHOLD;
	int rem = s_m%THRESHOLD;
	hipStream_t stream[sections];
	
	for(i = 0; i < sections; i++){
		hipStreamCreate(&stream[i]);	
	}
	printf("moving on...\n");

	hipDeviceSynchronize();

    	// Allocate CUDA events that we'll use for timing
    	hipEvent_t start;
    	error = hipEventCreate(&start);

    	if (error != hipSuccess)
    	{
        	fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        	exit(EXIT_FAILURE);
    	}

    	hipEvent_t stop;
    	error = hipEventCreate(&stop);

    	if (error != hipSuccess)
    	{
        	fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        	exit(EXIT_FAILURE);
    	}

    	// Record the start event
    	error = hipEventRecord(start, NULL);

    	if (error != hipSuccess)
    	{
        	fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
        	exit(EXIT_FAILURE);
    	}

	/* Parallel on Device */
 	/* Launch encrypt() kernel on GPU with N blocks */
	for(i = 0; i < sections-1; i++ ){
 		encrypt<<<(s_m+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(s_m, d_m, d_k, d_c);
		hipStreamSynchronize(stream[i]);
	}
 	
	// Record the stop event
    	error = hipEventRecord(stop, NULL);

    	if (error != hipSuccess)
    	{
        	fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        	exit(EXIT_FAILURE);
    	}

    	// Wait for the stop event to complete
    	error = hipEventSynchronize(stop);

    	if (error != hipSuccess)
    	{
        	fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        	exit(EXIT_FAILURE);
    	}

    	//float msecTotal = 0.0f;
  
	/* Copy result back to host */
 	error = hipMemcpy(c, d_c, s_m, hipMemcpyDeviceToHost);

	if (error != hipSuccess)
    	{
        	printf("hipMemcpy (c,d_c) returned error code %d, line(%d)\n", error, __LINE__);
        	exit(EXIT_FAILURE);
    	}

	error = hipEventElapsedTime(&msecTotal, start, stop);

	if (error != hipSuccess)
    	{
        	fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        	exit(EXIT_FAILURE);
    	}

	msecPerencrypt = msecTotal / 1;
    	printf( "Performance= %.06f sec\n", msecPerencrypt/1000.0 );

	//validate for correctness
 	for (j = 0; j < s_m; j++){
		if( c[j] != (m[j]^k[j]) ){
			printf("WRONG! c[%d] != m[%d]^k[%d] ==> c='%c',m^k=%c\n", j,j,j,c[j],m[j]^k[j]);
			//exit(EXIT_FAILURE);
		}
	}

	// Compute and print the performance
    	//float msecPerencrypt = msecTotal / 1;
	
	/* Cleanup */
	/* Destroy streams */
	//for (j = 0; j < sections; j++){
    		//cudaStreamDestroy(stream[j]);
  	//}

 	free(m); free(k); free(c);
 	hipFree(d_m); hipFree(d_k); hipFree(d_c);
 	fclose(fp_m); fclose(fp_k);

	return 0;
}
