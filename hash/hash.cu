/**
 * One Way Hash with CUDA (Fall 2016):
 * 
 * Members:
 * Emanuelle Crespi, Tolga Keskinoglu
 *
 * This test implements a simple hash from a space of size 2n --> n 
 *
 * The following code makes use of the kernel call hash(char *f, char *h, int n)
 * to perform a parallel hash of elements f --> h with corresponding indices 2i --> i 
 *
 * The result is a mapping of the data within f to the data within h 
 * The output is verified before the program terminates to see that every 
 * element at index 2i of f is indeed at index i in h
 *
 * We can see that there is a significant speedup in comparison to the time it takes
 * to perform the hash in the serial code.
 * 
 * The output of the performance is displayed in seconds. 
 * The performance results are to be compared with the performance of hash.c
 *
 */

// System includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>


// Jetson TK1 has device capability 1.x allowing 1024 threads/block
// We also indicate EVEN_NUM as the vector size since this hash requires even length arrays
#define THREADS_PER_BLOCK 1024 
#define EVEN_NUM 123374234


__global__ void hash(char *f, char *h, int n) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;

	if( i < n ){
		h[i] = f[2*i];	
	}
}

int main(void) {
	int two_n = EVEN_NUM, i, r=50;
	char *f, *h, *d_f,*d_h;
	hipError_t error;

	if ( two_n % 2 ){
		printf("NO NO NO!!! Even numbers only please.\n");
		exit(EXIT_FAILURE);
	}

	//printf("Malloc space on CPU (f,h)");
	f = (char *)calloc(sizeof(char), two_n);

	if( f == NULL ){
		fprintf(stderr,"Failed to allocate %d bytes for f.",two_n);
		exit(EXIT_FAILURE);
	}

	h = (char *)calloc(sizeof(char), two_n/2);

	if( h == NULL ){
		fprintf(stderr,"Failed to allocate %d bytes for h.",two_n/2);
		exit(EXIT_FAILURE);
	}

	/* Identify our streams */
	//printf("Malloc space on GPU (d_f,d_h)\n");
	error = hipMalloc((void **)&d_f, sizeof(char) * two_n);
	
	if( error != hipSuccess ){
		fprintf(stderr,"Failed to hipMalloc %d bytes for d_f.",two_n);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void **)&d_h, sizeof(char) * two_n/2);
	
	if( error != hipSuccess ){
		fprintf(stderr,"Failed to hipMalloc %d bytes for d_h.",two_n/2);
		exit(EXIT_FAILURE);
	}

	//populate data into array
	//printf("Generate vectored data (Size=%d bytes)\n",two_n);
	for (i = 0; i < two_n; i++) {
		f[i] = (char) ((i % 94) + 33);
	}

	//send data over the bus
	//printf("Send data to GPU\n");
	error = hipMemcpy( d_f, f, two_n, hipMemcpyHostToDevice);
	
	if (error != hipSuccess)
    	{
        	printf("hipMemcpy (d_f,f) returned error code %d, line(%d)\n", error, __LINE__);
        	exit(EXIT_FAILURE);
    	}

	/*************************** Setup for testing ************************************/
	//printf("Run kernel code \n");


	hipDeviceSynchronize();

    	// Allocate CUDA events that we'll use for timing
    	hipEvent_t start;
    	error = hipEventCreate(&start);

    	if (error != hipSuccess)
    	{
        	fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        	exit(EXIT_FAILURE);
    	}

    	hipEvent_t stop;
    	error = hipEventCreate(&stop);

    	if (error != hipSuccess)
    	{
        	fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        	exit(EXIT_FAILURE);
    	}
 
    	// Record the start event
    	error = hipEventRecord(start, NULL);

    	if (error != hipSuccess)
    	{
        	fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
        	exit(EXIT_FAILURE);
    	}
	printf("Running...\n");
	//run kernel
	for( i = 0; i < r; i++){
		hash<<<(two_n/2+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_f,d_h,two_n/2);
	}
	
	// Record the stop event
    	error = hipEventRecord(stop, NULL);

    	if (error != hipSuccess)
    	{
        	fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        	exit(EXIT_FAILURE);
    	}

    	// Wait for the stop event to complete
    	error = hipEventSynchronize(stop);

    	if (error != hipSuccess)
    	{
        	fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        	exit(EXIT_FAILURE);
    	}

    	float msecTotal = 0.0f;
    	error = hipEventElapsedTime(&msecTotal, start, stop);

    	if (error != hipSuccess)
    	{
        	fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        	exit(EXIT_FAILURE);
    	}

    	
	/*******************************************************************************************
	 ****************************** for testing purposes ***************************************/

	//send data over the bus
	error = hipMemcpy( h, d_h, sizeof(char)*two_n/2, hipMemcpyDeviceToHost);
	
	if (error != hipSuccess)
    	{
        	printf("hipMemcpy (h,d_h) returned error code %d, line(%d)\n", error, __LINE__);
        	exit(EXIT_FAILURE);
    	}
	//printf("Done.\n");

	//validate for correctness
	for (i = 0; i < two_n/2; i++) {
		if (h[i] != f[2*i]) {
			//printf("index %d FAILED!\n", i);
			exit(EXIT_FAILURE);
		}
	}

	// Compute and print the performance
    	float msecPerhash = msecTotal / 1;
    	printf( "Performance= %.06f sec\n", msecPerhash/1000.0 );

	free(f); free(h);
	hipFree(d_f); hipFree(d_h);
	
	hipDeviceReset();

	return 0;
}
